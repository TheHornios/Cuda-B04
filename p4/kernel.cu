﻿/**
* ARQUITECTURA DE COMPUTADORES
* 2º Grado en Ingenieria Informatica
*
* Básico 4
*
* Alumno: Rodrigo Pascual Arnaiz
* Fecha: 13/10/2022
*
*/

///////////////////////////////////////////////////////////////////////////
// includes
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h> 

///////////////////////////////////////////////////////////////////////////
// defines
#define M 6
#define N 21

///////////////////////////////////////////////////////////////////////////
// declaracion de funciones
// HOST: funcion llamada desde el host y ejecutada en el host
/**
* Funcion: propiedadesDispositivo
* Objetivo: Mustra las propiedades del dispositvo, esta funcion
* es ejecutada llamada y ejecutada desde el host
*
* Param: INT id_dispositivo -> ID del dispotivo
* Return: cudaDeviceProp -> retorna el onjeto que tiene todas las
* propiedades del dispositivo CUDA
*/
__host__ void propiedadesDispositivo(int id_dispositivo)
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, id_dispositivo);
	// calculo del numero de cores (SP)
	int cuda_cores = 0;
	int multi_processor_count = deviceProp.multiProcessorCount;
	int major = deviceProp.major;
	int minor = deviceProp.minor;
	char* arquitectura = (char*)"";

	switch (major)
	{
	case 1:
		//TESLA
		cuda_cores = 8;
		arquitectura = (char*)"TESLA";
		break;
	case 2:
		//FERMI
		arquitectura = (char*)"FERMI";
		if (minor == 0)
			cuda_cores = 32;
		else
			cuda_cores = 48;
		break;
	case 3:
		//KEPLER
		arquitectura = (char*)"KEPLER";
		cuda_cores = 192; 
		break;
	case 5:
		//MAXWELL
		arquitectura = (char*)"MAXWELL";
		cuda_cores = 128;
		break;
	case 6:
		//PASCAL
		arquitectura = (char*)"PASCAL";
		cuda_cores = 64;
		break;
	case 7:
		//VOLTA
		arquitectura = (char*)"VOLTA";
		cuda_cores = 64;
		break;
	case 8:
		//AMPERE
		arquitectura = (char*)"AMPERE";
		cuda_cores = 128;
		break;
	default:
		arquitectura = (char*)"DESCONOCIDA";
		//DESCONOCIDA
		cuda_cores = 0;
		printf("!!!!!dispositivo desconocido!!!!!\n");
	}
	// presentacion de propiedades
	printf("***************************************************\n");
	printf("DEVICE %d: %s\n", id_dispositivo, deviceProp.name);
	printf("***************************************************\n");
	printf("> Capacidad de Computo \t\t\t: %d.%d\n", major, minor);
	printf("> Arquitectura CUDA \t\t\t: %s \n", arquitectura);
	printf("> No. de MultiProcesadores \t\t: %d \n", multi_processor_count);
	printf("> No. de CUDA Cores (%dx%d) \t\t: %d \n", cuda_cores, multi_processor_count, cuda_cores *
		multi_processor_count);
	printf("> Memoria Global (total) \t\t: %zu MiB\n", deviceProp.totalGlobalMem / (1 << 20));
	printf("> No. max. de Hilos (por bloque) \t: %d \n",
		deviceProp.maxThreadsPerBlock);
	printf("***************************************************\n");
	printf("> KERNEL DE %i BLOQUE con %i HILOS:\n", 1, N * M);
	printf("\teje x -> %i hilos\n", N);
	printf("\teje y -> %i hilos\n", M);
}

///////////////////////////////////////////////////////////////////////////
// KERNEL: Función que deja las columnas impares a 0
/**
* Funcion: imparesCero
* Objetivo: Funcion que rellena un array pasado por parametro
* con los datos de otro array pasado por parametro pero dejando las celdas impares a 0
*
* Param: INT* original -> Puntero del array que tiene los datos 
* Param: INT* resultado -> Puntero del array a rellenar
* Return: void
*/
__global__ void imparesCero(int* original, int* resultado)
{
	// indice de fila
	int fila = threadIdx.y;
	// indice de columna
	int columna = threadIdx.x;
	// Calcular posición real
	int index = fila * N + columna;
	resultado[index] = columna % 2 != 0 ? 0 : original[index];
}


///////////////////////////////////////////////////////////////////////////
// HOST: funcion llamada desde el host y ejecutada en el host
/**
* Funcion: rellenarVectorAleatorio
* Objetivo: Funcion que rellena un array pasado por parametro
* con numero aleatorios del 1 al 9
*
* Param: INT* arr -> Puntero del array a rellenar
* Param: INT size -> Longitud del array
* Return: void
*/
__host__ void rellenarVectorAleatorio(int* arr, int size)
{
	for (size_t i = 0; i < size; i++)
	{
		arr[i] = ( rand() % 8 ) + 1;
	}
}

///////////////////////////////////////////////////////////////////////////
// MAIN: rutina principal ejecutada en el host
int main(int argc, char** argv)
{
	srand(time(NULL));
	// buscando dispositivos
	int numero_dispositivos;
	hipGetDeviceCount(&numero_dispositivos);
	// Guardar propiedades
	if (numero_dispositivos == 0)
	{
		printf("!!!!!ERROR!!!!!\n");
		printf("Este ordenador no tiene dispositivo de ejecucion CUDA\n");
		printf("<pulsa [INTRO] para finalizar>");
		getchar();
		return 1;
	}
	else
	{
		printf("Se han encontrado <%d> dispositivos CUDA:\n", numero_dispositivos);
		for (int id = 0; id < numero_dispositivos; id++)
		{
			propiedadesDispositivo(id);
		}
	}

	// Básico 4
	// Declaración de variables
	int* hst_original, * hst_final;
	int* dev_original, * dev_final;

	// reserva de memoria en el host
	hst_original = (int*)malloc(N * M * sizeof(int));
	hst_final = (int*)malloc(N * M * sizeof(int));

	// reserva de memoria en el device
	hipMalloc((void**)&dev_original, N * M * sizeof(int));
	hipMalloc((void**)&dev_final, N * M * sizeof(int));

	// Rellenar con numeros aleatorios, el array de dos direciones 
	rellenarVectorAleatorio(hst_original, ( M * N ) );

	// Copiar datos al dispositivo
	hipMemcpy( dev_original, hst_original, N * M * sizeof(int), hipMemcpyHostToDevice );

	// Dejar columnas impares a 0
	dim3 bloques(1);
	dim3 hilos(N, M);
	imparesCero <<<bloques, hilos >>> ( dev_original, dev_final );

	// Traer datos del device
	hipMemcpy(hst_final, dev_final, N * M * sizeof(int),hipMemcpyDeviceToHost);

	// Mostrar original y resultado 
	printf("> MATRIZ ORIGINAL:\n");
	for (int y = 0; y < M; y++)
	{
		for (int x = 0; x < N; x++)
		{
			printf("%i  ", hst_original[N * y + x]);
		}
		printf("\n");
	}
	printf("\n");
	printf("> MATRIZ FINAL:\n");
	for (int y = 0; y < M; y++)
	{
		for (int x = 0; x < N; x++)
		{
			printf("%i  ", hst_final[N * y + x]);
		}
		printf("\n");
	}
	printf("\n");
	// Salida del programa
	time_t fecha;
	time(&fecha);
	printf("***************************************************\n");
	printf("Programa ejecutado el: %s\n", ctime(&fecha));
	printf("<pulsa [INTRO] para finalizar>");
	getchar();
	return 0;
}
///////////////////////////////////////////////////////////////////////////